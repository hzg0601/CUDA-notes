
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello World from the the GPU\n");
}


int main(void)
{
    hello_from_gpu<<<4, 4>>>();
    hipDeviceSynchronize();

    return 0;
}
