#include "hip/hip_runtime.h"
/*********************************************************************************************
 * file name  : matrixSum1D_GPU_if_device.cu
 * author     : 权 双
 * date       : 2023-08-04
 * brief      : 矩阵求和程序，通过调用核函数在GPU执行
 *              添加：1、if判断条件；2、调用设备函数
***********************************************************************************************/

#include <stdio.h>
#include "../tools/common.cuh"

__device__ float add(const float x, const float y)
{
    return x + y;
}

__global__ void addFromGPU(float *A, float *B, float *C, const int N)
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int id = tid + bid * blockDim.x; // 513  32*17=544

    if (id >= N) return;
    C[id] = add(A[id], B[id]);
    
}


void initialData(float *addr, int elemCount)
{
    for (int i = 0; i < elemCount; i++)
    {
        addr[i] = (float)(rand() & 0xFF) / 10.f;
    }
    return;
}


int main(void)
{
    // 1、设置GPU设备
    setGPU();

    // 2、分配主机内存和设备内存，并初始化
    int iElemCount = 513;                     // 设置元素数量
    size_t stBytesCount = iElemCount * sizeof(float); // 字节数
    
    // （1）分配主机内存，并初始化
    float *fpHost_A, *fpHost_B, *fpHost_C;
    fpHost_A = (float *)malloc(stBytesCount);
    fpHost_B = (float *)malloc(stBytesCount);
    fpHost_C = (float *)malloc(stBytesCount);
    if (fpHost_A != NULL && fpHost_B != NULL && fpHost_C != NULL)
    {
        memset(fpHost_A, 0, stBytesCount);  // 主机内存初始化为0
        memset(fpHost_B, 0, stBytesCount);
        memset(fpHost_C, 0, stBytesCount);
    
    }
    else
    {
        printf("Fail to allocate host memory!\n");
        exit(-1);
    }


    // （2）分配设备内存，并初始化
    float *fpDevice_A, *fpDevice_B, *fpDevice_C;
    hipMalloc((float**)&fpDevice_A, stBytesCount);
    hipMalloc((float**)&fpDevice_B, stBytesCount);
    hipMalloc((float**)&fpDevice_C, stBytesCount);
    if (fpDevice_A != NULL && fpDevice_B != NULL && fpDevice_C != NULL)
    {
        hipMemset(fpDevice_A, 0, stBytesCount);  // 设备内存初始化为0
        hipMemset(fpDevice_B, 0, stBytesCount);
        hipMemset(fpDevice_C, 0, stBytesCount);
    }
    else
    {
        printf("fail to allocate memory\n");
        free(fpHost_A);
        free(fpHost_B);
        free(fpHost_C);
        exit(-1);
    }

    // 3、初始化主机中数据
    srand(666); // 设置随机种子
    initialData(fpHost_A, iElemCount);
    initialData(fpHost_B, iElemCount);
    
    // 4、数据从主机复制到设备
    hipMemcpy(fpDevice_A, fpHost_A, stBytesCount, hipMemcpyHostToDevice); 
    hipMemcpy(fpDevice_B, fpHost_B, stBytesCount, hipMemcpyHostToDevice); 
    hipMemcpy(fpDevice_C, fpHost_C, stBytesCount, hipMemcpyHostToDevice);


    // 5、调用核函数在设备中进行计算
    dim3 block(32);
    // dim3 grid(iElemCount / 32);  // 513 / 32 = 16
    dim3 grid((iElemCount + block.x - 1) / 32); //17

    addFromGPU<<<grid, block>>>(fpDevice_A, fpDevice_B, fpDevice_C, iElemCount);    // 调用核函数
    hipDeviceSynchronize();

    // 6、将计算得到的数据从设备传给主机
    hipMemcpy(fpHost_C, fpDevice_C, stBytesCount, hipMemcpyDeviceToHost);


    for (int i = 0; i < 10; i++)    // 打印
    {
        printf("idx=%2d\tmatrix_A:%.2f\tmatrix_B:%.2f\tresult=%.2f\n", i+1, fpHost_A[i], fpHost_B[i], fpHost_C[i]);
    }

    // 7、释放主机与设备内存
    free(fpHost_A);
    free(fpHost_B);
    free(fpHost_C);
    hipFree(fpDevice_A);
    hipFree(fpDevice_B);
    hipFree(fpDevice_C);

    hipDeviceReset();
    return 0;
}

