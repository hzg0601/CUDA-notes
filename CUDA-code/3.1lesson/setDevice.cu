
#include <hip/hip_runtime.h>
#include <stdio.h>


int main(void)
{
    // 检测计算机GPU数量
    int iDeviceCount = 0;
    hipError_t error = hipGetDeviceCount(&iDeviceCount);

    if (error != hipSuccess || iDeviceCount == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        exit(-1);
    }
    else
    {
        printf("The count of GPUs is %d.\n", iDeviceCount);
    }
    
    // 设置执行
    int iDev = 0;
    error = hipSetDevice(iDev);
    if (error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing.\n");
        exit(-1);
    }
    else
    {
        printf("set GPU 0 for computing.\n");
    }

    return 0;
}

