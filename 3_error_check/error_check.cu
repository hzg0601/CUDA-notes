#include <stdio.h>
#include <stdlib.h>
#include "../tools/common.cuh"

int main(void){
    float *host_A;
    int size_bytes = sizeof(float) * 2;
    host_A = (float *)malloc(size_bytes);
    memset(host_A, 0, size_bytes);

    float * deivce_A;
    hipError_t error_code = cuda_error_check(hipMalloc((float**) &deivce_A, 4),__FILE__, __LINE__);
    hipMemset(deivce_A, 0, size_bytes);

    cuda_error_check(hipMemcpy(deivce_A, host_A, size_bytes, hipMemcpyHostToDevice),__FILE__, __LINE__);

    free(host_A);
    cuda_error_check(hipFree(deivce_A),__FILE__,__LINE__);
    cuda_error_check(hipDeviceReset(),__FILE__, __LINE__);
    return 0;


}