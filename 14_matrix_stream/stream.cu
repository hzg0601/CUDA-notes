#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

#define BLOCK_SIZE 256
#define GRID_SIZE ((N+BLOCK_SIZE-1)/BLOCK_SIZE)
#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int *a,int *b, int *c, int n){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < n){
        int idx1 = (idx+1)%256;
        int idx2 = (idx+2)%256;
        float as = (a[idx] + a[idx1] + a[idx2])/3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2])/3.0f;
        c[idx] = (as + bs)/2.0f;
    }
}

void init_data(int * a, int n){
    srand(666);
    for(int i=0;i<n;i++) a[i] = rand();
}


int main(){
    // 检查设备是否支持deviceOverlap属性
    hipDeviceProp_t prop;
    int device_id;
    hipGetDevice(&device_id);
    hipGetDeviceProperties(&prop, device_id);
    if (!prop.deviceOverlap){
        printf("device id %d do not support deviceOverlap\n",device_id);
        exit(-1);
    }

    // 定义host变量,分配host锁页内存，初始化
    int * host_a, *host_b, *host_c;
    // host_a = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
    // host_b = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
    // host_c = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
    // 使用stream需使用hipHostAlloc函数定义host locked memory
    hipHostAlloc((int **)&host_a, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((int **)&host_b, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((int **)&host_c, FULL_DATA_SIZE*sizeof(int), hipHostMallocDefault);

    init_data(host_a,FULL_DATA_SIZE);
    init_data(host_b,FULL_DATA_SIZE);
    // 定义各个流的device变量，分配内存；
    int *dev_a0,*dev_a1,*dev_b0,*dev_b1,*dev_c0,*dev_c1;
    hipMalloc((int **)&dev_a0, sizeof(int)*N);
    hipMalloc((int **)&dev_a1, sizeof(int)*N);
    hipMalloc((int **)&dev_b0, sizeof(int)*N);
    hipMalloc((int **)&dev_b1, sizeof(int)*N);
    hipMalloc((int **)&dev_c0, sizeof(int)*N);
    hipMalloc((int **)&dev_c1, sizeof(int)*N);
    //声明并创建hipEvent_t和hipStream_t；
    hipEvent_t start, end;
    hipStream_t stream0,stream1;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    // 定义BLOCK和GRID
    dim3 block_size(BLOCK_SIZE);
    dim3 grid_size(GRID_SIZE);

    hipEventRecord(start);
    // 运行10次kernel,stream0和stream1交替执行,host_a占据前半部分,host_b占据后半部分，长度均为N
    for(int i=0; i<FULL_DATA_SIZE;i+=2*N){
        // 执行异步的数据拷贝
        hipMemcpyAsync(dev_a0, host_a+i, sizeof(int)*N, hipMemcpyHostToDevice,stream0);
        hipMemcpyAsync(dev_a1, host_a+i+N, sizeof(int)*N, hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_b0, host_b+i, sizeof(int)*N, hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b1, host_b+i+N, sizeof(int)*N, hipMemcpyHostToDevice, stream1);
        //调用核函数
        kernel<<<grid_size, block_size,0,stream0>>>(dev_a0, dev_b0, dev_c0, N);
        kernel<<<grid_size, block_size,0,stream1>>>(dev_a1, dev_b1, dev_c1, N);
        //流内回传数据
        hipMemcpyAsync(host_c+i,   dev_c0, sizeof(int)*N, hipMemcpyDeviceToHost,stream0);
        hipMemcpyAsync(host_c+i+N, dev_c1, sizeof(int)*N, hipMemcpyDeviceToHost,stream1);

    }
    // !流内同步在操作定义完成后执行
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    // 记录执行时间
    hipEventRecord(end);
    //!同步事件
    hipEventSynchronize(end);
    float elapse = 0.0;
    hipEventElapsedTime(&elapse, start, end);
    printf("time elapse:%f  ms\n",elapse);
    //销毁流和事件
    hipEventDestroy(start);
    hipEventDestroy(end);
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);

    //! 释放所有内存
    // !!!注意这里的内存由于使用hipHostAlloc分配的锁页内存，要用hipHostFree释放；
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a0);
    hipFree(dev_a1);
    hipFree(dev_b0);
    hipFree(dev_b1);
    hipFree(dev_c0);
    hipFree(dev_c1);
    return 0;

}