
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdint.h>
#include<time.h>     //for time()
#include<stdlib.h>   //for srand()/rand()
#include<sys/time.h> //for gettimeofday()/struct timeval
#include<assert.h>
	
#define KEN_CHECK(r) \
{\
    hipError_t rr = r;   \
    if (rr != hipSuccess)\
    {\
        fprintf(stderr, "CUDA Error %s, function: %s, line: %d\n",       \
		        hipGetErrorString(rr), __FUNCTION__, __LINE__); \
        exit(-1);\
    }\
}

#define M 3001  //three thousand and one nights
#define TILE_SIZE 32
__managed__ int shark[M][M];      //input matrix
__managed__ int gpu_shark_T[M][M];//GPU result
__managed__ int cpu_shark_T[M][M];//CPU result


__global__ void _ZHI_transpose(int A[M][M], int B[M][M])
{
    __shared__ int rafa[TILE_SIZE][TILE_SIZE + 1]; //tell me why?
	
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    if (col < M && row < M)
    {
	    rafa[threadIdx.y][threadIdx.x] = A[row][col];
    }
    __syncthreads();
	
    int y2 = threadIdx.y + blockDim.x * blockIdx.x;
    int x2 = threadIdx.x + blockDim.y * blockIdx.y;
    if (x2 < M && y2 < M)
    {
	    B[y2][x2] = rafa[threadIdx.x][threadIdx.y];
    }
}

void _sparks_transpose_cpu(int A[M][M], int B[M][M])
{
    for (int j = 0; j < M; j++)
    {
	for (int i = 0; i < M; i++)
	{
	    B[i][j] = A[j][i];
	}
    }
}

void DDBDDH_init(int A[M][M])
{
    uint32_t seed = (uint32_t)time(NULL); //make huan happy
    srand(seed);  //reseeding the random generator

    //filling the matrix with random data
    for (int j = 0; j < M; j++)
    {
	for (int i = 0; i < M; i++)
	{
	    A[j][i] = rand();
	}
    }
}

double get_time()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return ((double)tv.tv_usec * 0.000001 + tv.tv_sec);
}

int main()
{
    //**********************************
    fprintf(stderr, "DDBDDH is filling the %dx%d maxtrix with random data\n",
	            M, M);
    DDBDDH_init(shark);

    //**********************************
    //Now we are going to kick start your kernel.
    hipDeviceSynchronize(); //steady! ready! go!
    //Good luck & have fun!
    
    fprintf(stderr, "Running on GPU...\n");
    
double t0 = get_time();
    int n = (M + TILE_SIZE - 1) / TILE_SIZE; //what the hell is this!
    dim3 grid_shape(n, n);
    dim3 block_shape(TILE_SIZE, TILE_SIZE);
    _ZHI_transpose<<<grid_shape, block_shape>>>(shark, gpu_shark_T);
        KEN_CHECK(hipGetLastError());  //checking for launch failures
    KEN_CHECK(hipDeviceSynchronize()); //checking for run-time failurs
double t1 = get_time();

    //**********************************
    //Now we are going to exercise your CPU...
    fprintf(stderr, "Running on CPU...\n");

double t2 = get_time();
    _sparks_transpose_cpu(shark, cpu_shark_T);
double t3 = get_time();

    //******The last judgement**********
    for (int j = 0; j < M; j++)
    {
	for (int i = 0; i < M; i++)
	{
	    if (gpu_shark_T[j][i] != cpu_shark_T[j][i])
	    {
	        fprintf(stderr, "Test failed!\n");
	   	exit(-1);
	    }
	}
    }	
    fprintf(stderr, "Test Passed!\n");
    
    //****and some timing details*******
    fprintf(stderr, "GPU time %.3f ms\n", (t1 - t0) * 1000.0);
    fprintf(stderr, "CPU time %.3f ms\n", (t3 - t2) * 1000.0);

    return 0;
}	
	
