
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdint.h>
#include<time.h>     //for time()
#include<stdlib.h>   //for srand()/rand()
#include<sys/time.h> //for gettimeofday()/struct timeval


#define KEN_CHECK(r) \
{\
    hipError_t rr = r;   \
    if (rr != hipSuccess)\
    {\
        fprintf(stderr, "CUDA Error %s, function: %s, line: %d\n",       \
		        hipGetErrorString(rr), __FUNCTION__, __LINE__); \
        exit(-1);\
    }\
}

#define N 10000000
#define BLOCK_SIZE 256
#define BLOCKS ((N + BLOCK_SIZE - 1) / BLOCK_SIZE) //try next line if you can
//#define BLOCKS 666

__managed__ int source[N];               //input data
__managed__ int _partial_results[BLOCKS];//for 2-pass kernel
__managed__ int final_result[1] = {0};   //scalar output


__global__ void _hawk_sum_gpu(int *input, int count, int *output)
{
    __shared__ int bowman[BLOCK_SIZE];

    //**********register summation stage***********
    int komorebi = 0;
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
         idx < count;
	 idx += gridDim.x * blockDim.x
	)
    {
        komorebi += input[idx];
    }

    bowman[threadIdx.x] = komorebi;  //the per-thread partial sum is komorebi!
    __syncthreads();

    //**********shared memory summation stage***********
    for (int length = BLOCK_SIZE / 2; length >= 1; length /= 2)
    {
        int double_kill = -1;
	if (threadIdx.x < length)
	{
	    double_kill = bowman[threadIdx.x] + bowman[threadIdx.x + length];
	}
	__syncthreads();  //why we need two __syncthreads() here, and,
	
	if (threadIdx.x < length)
	{
	    bowman[threadIdx.x] = double_kill;
	}
	__syncthreads();  //....here ?
	
    } //the per-block partial sum is bowman[0]

    if (blockDim.x * blockIdx.x < count) //in case that our users are naughty
    {
        //per-block result written back, by thread 0, on behalf of a block.
        if (threadIdx.x == 0) output[blockIdx.x] = bowman[0];
    }
}

int _hawk_sum_cpu(int *ptr, int count)
{
    int sum = 0;
    for (int i = 0; i < count; i++)
    {
        sum += ptr[i];
    }
    return sum;
}

void _nanana_init(int *ptr, int count)
{
    uint32_t seed = (uint32_t)time(NULL); //make huan happy
    srand(seed);  //reseeding the random generator

    //filling the buffer with random data
    for (int i = 0; i < count; i++) ptr[i] = rand();
}

double get_time()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return ((double)tv.tv_usec * 0.000001 + tv.tv_sec);
}

int main()
{
    //**********************************
    fprintf(stderr, "nanana is filling the buffer with %d elements...\n", N);
    _nanana_init(source, N);

    //**********************************
    //Now we are going to kick start your kernel.
    hipDeviceSynchronize(); //steady! ready! go!
    //Good luck & have fun!
    
    fprintf(stderr, "Running on GPU...\n");
    
double t0 = get_time();
    _hawk_sum_gpu<<<BLOCKS, BLOCK_SIZE>>>(source, N, _partial_results);
        KEN_CHECK(hipGetLastError());  //checking for launch failures
	
    _hawk_sum_gpu<<<1, BLOCK_SIZE>>>(_partial_results, BLOCKS, final_result);
        KEN_CHECK(hipGetLastError());  //the same
	
    KEN_CHECK(hipDeviceSynchronize()); //checking for run-time failurs
double t1 = get_time();

    int A = final_result[0];
    fprintf(stderr, "GPU sum: %u\n", A);


    //**********************************
    //Now we are going to exercise your CPU...
    fprintf(stderr, "Running on CPU...\n");

double t2 = get_time();
    int B = _hawk_sum_cpu(source, N);
double t3 = get_time();
    fprintf(stderr, "CPU sum: %u\n", B);

    //******The last judgement**********
    if (A == B)
    {
        fprintf(stderr, "Test Passed!\n");
    }
    else
    {
        fprintf(stderr, "Test failed!\n");
	exit(-1);
    }
    
    //****and some timing details*******
    fprintf(stderr, "GPU time %.3f ms\n", (t1 - t0) * 1000.0);
    fprintf(stderr, "CPU time %.3f ms\n", (t3 - t2) * 1000.0);

    return 0;
}	
	
