#include "hip/hip_runtime.h"
#include <time.h>
#include <cmath>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
 
#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <stdio.h>

#include <iostream>
#include <vector>

#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv; 

#define DEBUG

 
__global__ void kernel_fftshift2D(hipfftDoubleComplex *IM, int im_height, int im_width);
__global__ void componentwiseMatrixMul(hipfftDoubleComplex *in1, hipfftDoubleComplex *in2,hipfftDoubleComplex *out, int row, int col);
__global__ void zeroPadding(hipfftDoubleComplex* F, hipfftDoubleComplex* FP, int newCols, int newRows, int oldCols, int oldRows); 


__global__ void zeroPadding(hipfftDoubleComplex* F, hipfftDoubleComplex* FP, int newCols, int newRows, int oldCols, int oldRows)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int ind = idx*newCols + idy;
    
    if(idx < newRows && idy < newCols)
    {       
        if(idx <oldRows && idy < oldCols)
        {
            FP[ind].x = F[idx*oldCols+idy].x;
        }
        else if(idx>oldRows || idy>oldCols)
        {
            FP[ind].x=0;
        }
    }
}


__global__ void componentwiseMatrixMul(hipfftDoubleComplex *in1, hipfftDoubleComplex *in2,hipfftDoubleComplex *out,int row, int col)
{
    int indexRow=threadIdx.x + blockIdx.x*blockDim.x; 
    int indexCol=threadIdx.y + blockIdx.y*blockDim.y; 
    if(indexRow<row && indexCol<col)
    {   
        out[indexRow*col+indexCol].x = in1[indexRow*col+indexCol].x*in2[indexRow*col+indexCol].x;
        out[indexRow*col+indexCol].y = in1[indexRow*col+indexCol].y*in2[indexRow*col+indexCol].y;
    }
}


__global__ void kernel_fftshift2D(hipfftDoubleComplex *IM, int imH, int imW)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int ind = idy*imW + idx;
    int x, y, indshift;
    hipfftDoubleComplex v;


    if(idx < imW && idy < imH/2)
    {
        if(idx<imW/2 && idy<imH/2)
        {
            x=idx+imW/2;
            y=idy+imH/2;
        }
        else if(idx>=imW/2 && idy<imH/2)
        {
            x=idx-imW/2;
            y=idy+imH/2;
        }

        indshift = y*imW+x;
        v.x = IM[ind].x;
        v.y = IM[ind].y;

        IM[ind].x = IM[indshift].x;
        IM[ind].y = IM[indshift].y;

        IM[indshift].x = v.x;
        IM[indshift].y = v.y;
    }
}


/*
 *  Image Convolution with CUFFT
 *  Correct usage : ./filter <num_threads> <output_file_path> <image_path> <kernel_path>
 */
int main(int argc, char* argv[])
{
    hipfftDoubleComplex *IM;
    hipfftDoubleComplex *FK;
    hipfftDoubleComplex *IK;
    hipfftHandle planZ2Z, planIZ2Z, plan2Z2Z;
    hipfftResult cuError;      
    StopWatchInterface *timer=NULL;
    
    int im_height, im_width, dimK = 5, num_threads, nDevices;
    hipfftDoubleComplex *im_d,*im_K;
    float gpuTime;
    
    char * output_file_path, *image_path, *kernel_path;
    Mat img;
    
    if (argc < 5){
        printf("%s - correct usage: %s <num_threads> <output_file_path> <image_path> <kernel_path>\n Setting default values. . .\n",argv[0],argv[0]);    
        im_height = im_width = 512;
        num_threads = 32;
        output_file_path = (char *)"filtrata.txt";
        image_path = (char *)"512Gaussian-Noise.jpg";
        kernel_path = (char *)"Kernel51.txt";
    } else {
        image_path = argv[3];
        num_threads = atoi(argv[1]);
        output_file_path = argv[2];
        kernel_path = argv[4];
    }
    
    img = imread(image_path,0);
    if (!img.data)
    {
        printf("Could not open image \n");
        return 1;
    }
    im_width = img.cols;
    im_height = img.rows;
        
    printf("im_width:%d im_height:%d \t #threads:%d \t Output file: %s\n",im_width,im_height,num_threads,output_file_path);
    
    hipGetDeviceCount(&nDevices);
    char maxClockDevice = 0;
    int memoryClockRate = 0;
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
#ifdef DEBUG
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Max Thread per block: %d\n\n", prop.maxThreadsPerBlock);
#endif
        if(prop.memoryClockRate > memoryClockRate){
            memoryClockRate = prop.memoryClockRate;
            maxClockDevice = i;   
        }
    }
    // Setting the fastest GPU device
    hipSetDevice(maxClockDevice);

    printf("Set Device [%d] for Execution\n",maxClockDevice);

    dim3 dimBlock(num_threads, num_threads);
    int nbBlocksW = im_width/num_threads;
    if((im_width%num_threads) != 0)
        nbBlocksW++;
    int nbBlocksH = im_height/num_threads;
    if((im_height%num_threads) != 0)
        nbBlocksH++;
    dim3 dimGrid(nbBlocksW, nbBlocksH);
    

    // Opening kernel file and output file
    FILE *fd2,*fd1;
    if((fd2=fopen(kernel_path,"r"))==NULL)
    {
        printf("Can't read kernel.\n");
    }
    if((fd1=fopen(output_file_path,"w"))==NULL)
    {
        printf("Can't open output file.\n");
    } 

    hipfftDoubleComplex **A=new hipfftDoubleComplex*[im_height];
    img.convertTo(img,CV_8UC1);
    for (int i=0;i<im_height;i++)
    {
        A[i] = new hipfftDoubleComplex[im_width];
        for (int j=0;j<im_width;j++)
        {
            A[i][j].x=(double)img.at<uchar>(j,i);
            A[i][j].y=0;
            //printf("%lf\n",A[i][j].x);
        }
    }
    /* - - - Building the Kernel with 0-padding - - - */
    hipfftDoubleComplex **K=new hipfftDoubleComplex*[im_height];
    for (int i=0;i<im_height;i++)
    {
        double numk;
        K[i]=new hipfftDoubleComplex[im_width];
        for (int j=0;j<im_width;j++)
        {
            if((i >= ((im_height/2) - 2)) && (i <= ((im_height/2) + 2)) && (j >=((im_width/2)-2)) && (j <=((im_width/2)+2))){
                fscanf(fd2,"%lf",&numk);
                K[i][j].x=numk;
                K[i][j].y=0.0;
            } else {
                K[i][j].x=0.0;
                K[i][j].y=0.0;
            }
        }
    }
    /* ------------------------------    Memory allocation    --------------------------------- */
    if(hipMalloc((void**) &im_d, im_width*im_height*sizeof(hipfftDoubleComplex)) != hipSuccess){
        cout<<"Error Memory Allocation (line "<<__LINE__<<")"<<endl;
        return 0;
    }
    if(hipMalloc((void**) &im_K, dimK*dimK*sizeof(hipfftDoubleComplex)) != hipSuccess){
        cout<<"Error Memory Allocation (line "<<__LINE__<<")"<<endl;
        return 0;
    }
    if(hipMalloc((void**) &IM, im_width*im_height*sizeof(hipfftDoubleComplex)) != hipSuccess)
    {
        cout<<"Error Memory Allocation (line "<<__LINE__<<")"<<endl;
        return 0;
    }
    if(hipMalloc((void**) &IK, im_width*im_height*sizeof(hipfftDoubleComplex)) != hipSuccess)
    {
        cout<<"Error Memory Allocation (line "<<__LINE__<<")"<<endl;
        return 0;
    }
    if(hipMalloc((void**) &FK, im_width*im_height*sizeof(hipfftDoubleComplex)) != hipSuccess)
    {
        cout<<"Error Memory Allocation (line "<<__LINE__<<")"<<endl;
        return 0;
    }

    // Creating and starting timer
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);  

    /* --- Copying image and kernel on device --- */
    for(int i=0; i<im_height; ++i)
    {
        hipMemcpy2D(im_d + i*im_width, sizeof(hipfftDoubleComplex), A[i], sizeof(hipfftDoubleComplex), sizeof(hipfftDoubleComplex), im_width, hipMemcpyHostToDevice);
    }
    
    for(int i=0; i<im_height; ++i)
    {
        hipMemcpy2D(IK + i*im_width, sizeof(hipfftDoubleComplex), K[i], sizeof(hipfftDoubleComplex), sizeof(hipfftDoubleComplex), im_width, hipMemcpyHostToDevice);
    }
    
    /* Creating plans */
    cuError = hipfftPlan2d(&planZ2Z, im_width, im_height, HIPFFT_Z2Z);
    if(cuError != HIPFFT_SUCCESS)
    {
        cout<<"Error creating FFT plan (line "<<__LINE__<<")"<<endl;
        return 0;
    }
    cuError = hipfftPlan2d(&plan2Z2Z, im_width, im_height, HIPFFT_Z2Z);
    if(cuError != HIPFFT_SUCCESS)
    {
        cout<<"Error creating FFT plan (line "<<__LINE__<<")"<<endl;
        return 0;
    }
    cuError = hipfftPlan2d(&planIZ2Z, im_width, im_height, HIPFFT_Z2Z);
    if(cuError != HIPFFT_SUCCESS)
    {
        cout<<"Error creating IFFT plan (line "<<__LINE__<<")"<<endl;
        return 0;
    }
    
    /* - - - Fast Fourier Transform on image - - - */
    cuError = hipfftExecZ2Z(planZ2Z,im_d, IM, HIPFFT_FORWARD);
    if(cuError != HIPFFT_SUCCESS)
    {
        cout<<"Error "<<cuError<<" during executing CUFFT (line "<<__LINE__<<")"<<endl;
        return cuError;
    }
    kernel_fftshift2D<<<dimGrid, dimBlock>>>(IM, im_height, im_width);
    
    /* - - - Fast Fourier Transform on kernel - - - */
    cuError=hipfftExecZ2Z(plan2Z2Z,IK, FK,HIPFFT_FORWARD);
    if(cuError != HIPFFT_SUCCESS)
    {
        cout<<"Error "<<cuError<<" during executing CUFFT (line "<<__LINE__<<")"<<endl;
        return cuError;
    }
    kernel_fftshift2D<<<dimGrid, dimBlock>>>(FK, im_height, im_width);
    
    /* Component-wise matrix-mul */
    componentwiseMatrixMul<<<dimGrid, dimBlock>>> (IM,FK,IM, im_height, im_width);
    
    /* - - - Executing IFFT and shifting back - - - */
#ifdef DEBUG
    printf("Shifted Image\n");    
#endif    
    kernel_fftshift2D<<<dimGrid, dimBlock>>>(IM, im_height, im_width);
    
    cuError=hipfftExecZ2Z(planIZ2Z, IM,im_d,HIPFFT_BACKWARD);
    if(cuError != HIPFFT_SUCCESS)
    {
        cout<<"Error "<<cuError<<" during executing CUIFFT (line "<<__LINE__<<")"<<endl;
        return cuError;
    }
    kernel_fftshift2D<<<dimGrid, dimBlock>>>(im_d, im_height, im_width);
    
    /* - - - Generating output - - - */
    hipfftDoubleComplex *c = (hipfftDoubleComplex*)malloc(im_width*im_height*sizeof(hipfftDoubleComplex));
    hipMemcpy(c, im_d, sizeof(hipfftDoubleComplex)*im_height*im_width , hipMemcpyDeviceToHost);
    
    // Stopping timer and computing elapsed time
    sdkStopTimer(&timer);
    gpuTime = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    
    printf("Execution time %8.4f ms\n",gpuTime); //Printing elapsed time
#ifdef DEBUG    
    printf("Generating output. . .\n");
#endif    
    long double max = c[0].x;
    for (int i = 0; i < im_height; i++)
    {
        for (int j =0 ; j< im_width; j++)
        {
            fprintf(fd1,"%lf ",c[i*im_width + j].x);
            if(c[i*im_width + j].x > max)
                max = c[i*im_width + j].x;
        }
        fprintf(fd1,"\n");
    }
    img.convertTo(img,CV_64F);
    for(int i = 0; i < im_height; i++){
        for(int j =0; j < im_width; j++){
            img.at<double>(j,i) = floor((c[i*im_width + j].x/max)*255);
            //printf("%d\n",(unsigned)floor((c[i*im_width + j].x/max)*255));
        }
    }
    imwrite("output_image.jpg",img);
 
    free(c);
    free(A);
    free(K);
    hipFree(im_d);
    hipFree(IM);
    hipFree(IK);
    hipFree(FK);
    hipfftDestroy(plan2Z2Z);
    hipfftDestroy(planIZ2Z);
    hipfftDestroy(planZ2Z);
    
    return 0;
}
